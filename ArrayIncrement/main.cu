
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void increment_gpu(int *a, int N) {
    int i = threadIdx.x;
    if (i < N) 
        a[i] += 1;
}

int main() {
    const int N = 5;
    int h_a[N] = {1, 2, 3, 4, 5};

    int *d_a;
    hipMalloc(&d_a, N * sizeof(int));
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid_size(1); 
    dim3 block_size(N);

    increment_gpu<<<grid_size, block_size>>>(d_a, N);

    hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%d ", h_a[i]);
    }
    printf("\n");

    hipDeviceSynchronize();

    free(h_a);
    hipFree(d_a);
}
