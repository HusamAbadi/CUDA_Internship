// ! Using cudaMalloc()


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <vector>
#include <algorithm>

__global__
void vecAddKernel(int* A, int* B, int* C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<n) C[i] = A[i] + B[i];
}

int main(){
    constexpr int N = 1 << 9;
    size_t bytes = N * sizeof(int);

    // Vectors for holding the host-side data
    std::vector<int> a(N);
    std::vector<int> b(N);
    std::vector<int> c(N);
    
    // Initializing the vectors with random numbers in each array between 0-100
    std::generate(begin(a), end(a), []() { return rand() % 100; });
    std::generate(begin(b), end(b), []() { return rand() % 100; });
    
    // Allocate memory on the GPU
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    
    // Copy data to the GPU
    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);
    
    // Thread per CTA (256 Threads)
    int threads = 1 << 8;
    // Blocks per Grid (1 Block)
    int blocks = (int)ceil(N / (float)threads);
    
    // Setup our kernel launch parameters
    dim3 block(threads, 1, 1);
    dim3 grid(blocks, 1, 1);
    
    // Launch our kernel
    vecAddKernel<<<grid, block>>>(d_a, d_b, d_c, N);
    
    // Copy data back to the host
    hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    // Display results on the terminal
    for(int i = 0; i < N; i++){
        std::cout << "a[" << i << "] + b[" << i << "] = " << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    }

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "Success!" << std::endl;

    return 0;
}