
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<n) C[i] = A[i] + B[i];
}

// Stub Function
void vecAdd(float* A, float* B, float* C, int n){
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_B, size);
    hipMalloc((void **) &d_C, size);
    
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    
    
    //Setup our kernel launch parameters
    int threads = 32;
    int blocks = (int)ceil(n / threads);

    //Launch Kernel
    vecAddKernel<<<blocks, threads>>>(d_A, d_B, d_C, n);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);


}


int main(){
    const int n = 1 << 5;
    // n = 32

    float a[n] = {47, 30, 14, 70, 21, 97, 65, 78, 61, 86, 2, 20, 88, 45, 80, 33, 94, 54, 24, 50, 68, 36, 83, 57, 79, 11, 7, 72, 69, 89, 40, 16};

    float b[n] = {23, 67, 91, 13, 25, 72, 89, 3, 98, 61, 11, 85, 31, 69, 45, 16, 87, 53, 44, 6, 96, 5, 29, 82, 59, 10, 93, 2, 95, 81, 33, 64};

        
    float c[n];

    // Calling the Stub Function
    vecAdd(a, b, c, n);

    for(int i = 0; i < n; i++){
        printf("%1.0f ", c[i]);
    }
    printf("\n");

    hipDeviceSynchronize();

}