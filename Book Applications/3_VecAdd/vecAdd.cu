
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<n) C[i] = A[i] + B[i];
}

// Stub Function
void vecAdd(float* A, float* B, float* C, int n){
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **) &d_C, size);
    
    //Kernel Invocation code
    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);


}


int main(){
    const int n = 5;
    float a[n] = {1, 2, 3, 4, 5};
    float b[n] = {10, 11, 12, 13, 14};
    float c[n];

    // Calling the Stub Function
    vecAdd(a, b, c, n);

    for(int i = 0; i < n; i++){
        printf("%1.0f ", c[i]);
    }
    hipDeviceSynchronize();

}