
#include <hip/hip_runtime.h>
#include <stdio.h>

void vecAdd(float* A, float* B, float* C, int n){
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    printf("Hi!");

    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **) &d_C, size);
    
    //Kernel Invocation code - to be shown later

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);

}


int main(){
    const int N = 5;
    float a[5] = {1, 2, 3, 4, 5};
    float b[5] = {5, 4, 3, 2, 1};
    float c[5];

    vecAdd(a, b, c, N);
}