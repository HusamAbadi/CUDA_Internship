
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void vectorAdd(const int *A, const int *B, int *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1<<20;
    size_t size = N * sizeof(int);

    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
    }

    int *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", h_A[i], h_B[i], h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
